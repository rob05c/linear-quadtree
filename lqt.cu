#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

#include "lqt.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>    
#include <linux/hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>

using namespace hipcub; // debug



/// \todo fix to not be global
CachingDeviceAllocator g_allocator(true); // CUB caching allocator for device memory

__global__ void cuda_cuda_nodify(struct lqt_point* points, location_t* locations,
                                 const size_t depth, ord_t xstart, ord_t xend, 
                                 ord_t ystart, ord_t yend, size_t len) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i >= len)
    return; // skip the final block remainder

  struct lqt_point* thisPoint = &points[i];

  ord_t currentXStart = xstart;
  ord_t currentXEnd = xend;
  ord_t currentYStart = ystart;
  ord_t currentYEnd = yend;
  for(size_t j = 0, jend = depth; j != jend; ++j) {
    const location_t bit1 = thisPoint->y > (currentYStart + (currentYEnd - currentYStart) / 2);
    const location_t bit2 = thisPoint->x > (currentXStart + (currentXEnd - currentXStart) / 2);
    const location_t currentPosBits = (bit1 << 1) | bit2;
    locations[i] = (locations[i] << 2) | currentPosBits;

    const ord_t newWidth = (currentXEnd - currentXStart) / 2;
    currentXStart = floor((thisPoint->x - currentXStart) / newWidth) * newWidth + currentXStart;
    currentXEnd = currentXStart + newWidth;
    const ord_t newHeight = (currentYEnd - currentYStart) / 2;
    currentYStart = floor((thisPoint->y - currentYStart) / newHeight) * newHeight + currentYStart;
    currentYEnd = currentYStart + newHeight;
  }
}

struct linear_quadtree cuda_nodify(struct lqt_point* points, size_t len, 
             ord_t xstart, ord_t xend, 
             ord_t ystart, ord_t yend,
             size_t* depth) {
  *depth = LINEAR_QUADTREE_DEPTH;

  const size_t THREADS_PER_BLOCK = 512;

  location_t*       cuda_locations;
  struct lqt_point* cuda_points;
  hipMalloc((void**)&cuda_locations, len * sizeof(location_t));
  hipMalloc((void**)&cuda_points, len * sizeof(struct lqt_point));
  hipMemcpy(cuda_points, points, len * sizeof(struct lqt_point), hipMemcpyHostToDevice);
  hipMemset(cuda_locations, 0, len * sizeof(location_t)); // debug
  fprintf(stderr, "cn calling cuda_nodify_nodify\n");
  cuda_cuda_nodify<<<(len + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(cuda_points, cuda_locations, *depth, xstart, xend, ystart, yend, len);
  fprintf(stderr, "cn cuda_nodify_nodify returned\n");
  fprintf(stderr, "cn locations malloc\n");
  location_t* locations = (location_t*) malloc(len * sizeof(location_t));
  fprintf(stderr, "cn locations memcpy\n");
  hipMemcpy(locations, cuda_locations, len * sizeof(location_t), hipMemcpyDeviceToHost);
  fprintf(stderr, "cn locations free\n");
  hipFree(cuda_locations);
  fprintf(stderr, "cn points free\n");
  hipFree(cuda_points);

  struct linear_quadtree lqt;
  lqt.points    = points;
  lqt.locations = locations;
  lqt.length    = len;
  return lqt;
}

void cuda_sortify(struct linear_quadtree lqt) {
  DoubleBuffer<location_t> d_keys;
  DoubleBuffer<lqt_point> d_values;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(location_t) * lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(location_t) * lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[0], sizeof(lqt_point) * lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(lqt_point) * lqt.length));

  size_t temp_storage_bytes = 0;
  void* d_temp_storage = NULL;

  CubDebugExit( hipMemcpy(d_keys.d_buffers[0], lqt.locations, sizeof(location_t) * lqt.length, hipMemcpyHostToDevice));
  CubDebugExit( hipMemcpy(d_values.d_buffers[0], lqt.points, sizeof(lqt_point) * lqt.length, hipMemcpyHostToDevice));
//  CubDebugExit( hipMemcpy(d_keys.d_buffers[1], lqt.locations, sizeof(location_t) * lqt.length, hipMemcpyHostToDevice));
//  CubDebugExit( hipMemcpy(d_values.d_buffers[1], lqt.points, sizeof(lqt_point) * lqt.length, hipMemcpyHostToDevice));

  // get size of temp storage
  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, lqt.length));
  CubDebugExit( g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  printf("CS temp size: %u\n", temp_storage_bytes);

  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, lqt.length));
  
  CubDebugExit( hipMemcpy(lqt.locations, d_keys.Current(), lqt.length * sizeof(location_t), hipMemcpyDeviceToHost));
  CubDebugExit( hipMemcpy(lqt.points, d_values.Current(), lqt.length * sizeof(lqt_point), hipMemcpyDeviceToHost));

  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_temp_storage));
}

void print_array_uint(unsigned int* array, const size_t len) {
  if(len == 0)
    return;
  printf("[%u", array[0]);
  for(size_t i = 1, end = len; i != end; ++i)
    printf(" %u", array[i]);
  printf("]");
}
void print_array_int(int* array, const size_t len) {
  if(len == 0)
    return;
  printf("[%d", array[0]);
  for(size_t i = 1, end = len; i != end; ++i)
    printf(" %d", array[i]);
  printf("]");
}

template <typename T> struct fmt_traits;
template <>
struct fmt_traits<int> {
  static const char* str() {return "%d";}
};
template <>
struct fmt_traits<unsigned int> {
  static const char* str() {return "%u";}
};
template <>
struct fmt_traits<location_t> {
  static const char* str() {return "%lu";}
};

template <typename T>
void print_array(T* array, const size_t len) {
  if(len == 0)
    return;
  printf("[");
  printf(fmt_traits<T>::str(), array[0]);
  for(size_t i = 1, end = len; i != end; ++i) {
    printf(" ");
    printf(fmt_traits<T>::str(), array[i]);
  }
  printf("]");
}


void test_cub() {
  typedef location_t key_t;
  typedef int value_t;
  const size_t len = 10;
  key_t* keys = new key_t[len];
  value_t* values = new value_t[len];
  for(size_t i = 0, end = len; i != end; ++i) {
    keys[i] = (key_t) rand() % 11;
    values[i] = i;
  }

  printf("unsorted:\n");
  print_array(keys, len);
  printf("\n");
  print_array(values, len);
  printf("\n");


  DoubleBuffer<key_t> d_keys;
  DoubleBuffer<value_t> d_values;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(key_t) * len));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(key_t) * len));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[0], sizeof(value_t) * len));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(value_t) * len));

  size_t temp_storage_bytes = 0;
  void* d_temp_storage = NULL;

  CubDebugExit( hipMemcpy(d_keys.d_buffers[0], keys, sizeof(key_t) * len, hipMemcpyHostToDevice));
  CubDebugExit( hipMemcpy(d_values.d_buffers[0], values, sizeof(value_t) * len, hipMemcpyHostToDevice));
//  CubDebugExit( hipMemcpy(d_keys.d_buffers[1], lqt.locations, sizeof(key_t) * len, hipMemcpyHostToDevice));
//  CubDebugExit( hipMemcpy(d_keys.d_buffers[1], lqt.povalue_ts, sizeof(value_t) * len, hipMemcpyHostToDevice));

  // get size of temp storage
  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, len));
  CubDebugExit( g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  printf("CS temp size: %u\n", temp_storage_bytes);

  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, len));
  
  CubDebugExit( hipMemcpy(keys, d_keys.Current(), len * sizeof(key_t), hipMemcpyDeviceToHost));
  CubDebugExit( hipMemcpy(values, d_values.Current(), len * sizeof(value_t), hipMemcpyDeviceToHost));

  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_temp_storage));


  printf("sorted:\n");
  print_array(keys, len);
  printf("\n");
  print_array(values, len);
  printf("\n");

  delete[] keys;
  delete[] values;
}

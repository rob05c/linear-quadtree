#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

#include "lqt.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>    
#include <linux/hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>

using namespace hipcub; // debug

/// \todo fix to not be global
CachingDeviceAllocator g_allocator(true); // CUB caching allocator for device memory

/// \returns the device totalGlobalMem
inline size_t GetDeviceMemory() {
  hipDeviceProp_t properties;
  int deviceNum;
  CubDebugExit(hipGetDevice(&deviceNum));
  CubDebugExit(hipGetDeviceProperties(&properties, deviceNum));
  return properties.totalGlobalMem;
}

#ifndef MIN
#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#endif

inline size_t find_min(location_t* keys, const size_t keys_len) {
  if(keys_len == 0)
    return 0;
  location_t min = keys[0];
  size_t min_key = 0;
  for(size_t i = 0, end = keys_len; i != end; ++i) {
    if(keys[i] < min) {
      min_key = i;
      min = keys[i];
    }
  }
  return min_key;
}

/// \param[out] keys must be at least block_len large
/// \return whether all iterators are past their length. That is, when this is false, we can stop merging.
inline bool get_keys(location_t* keys, const struct linear_quadtree* array_blocks, const size_t block_len, const size_t* iterators) {
  bool got_key = false;
  for(int i = 0, end = block_len; i != end; ++i) {
    if(iterators[i] >= array_blocks[i].length) {
      keys[i] = location_t_max; // we've iterated past this block's len; make sure this key is never the min.
      continue;
    }
    got_key = true;
    keys[i] = array_blocks[i].locations[iterators[i]];
  }
  return got_key;
}

struct linear_quadtree lqt_merge(struct linear_quadtree* array_blocks, const size_t block_len, struct lqt_point* points, const size_t len) {
  struct linear_quadtree lqt;
  lqt.points    = points;
  lqt.locations = (location_t*) malloc(sizeof(location_t) * len);
  lqt.length    = len;
  if(len == 0)
    return lqt;

  size_t lqt_iterator = 0;
  size_t* iterators = (size_t*) malloc(sizeof(size_t) * block_len);
  for(size_t i = 0, end = block_len; i != end; ++i)
    iterators[i] = 0;

  {
    location_t keys[block_len];  
    for(size_t i = 0; get_keys(keys, array_blocks, block_len, iterators); ++i) {
      const size_t min_block = find_min(keys, block_len);
      lqt.locations[lqt_iterator] = array_blocks[min_block].locations[iterators[min_block]];
      lqt.points[lqt_iterator]    = array_blocks[min_block].points[iterators[min_block]];
      ++iterators[min_block];
      ++lqt_iterator;
    }
  }
  
  free(iterators);
  return lqt;
}

__global__ void nodify_kernel(struct lqt_point* points, location_t* locations,
                                 const size_t depth, ord_t xstart, ord_t xend, 
                                 ord_t ystart, ord_t yend, size_t len) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i >= len)
    return; // skip the final block remainder

  struct lqt_point* thisPoint = &points[i];

  ord_t currentXStart = xstart;
  ord_t currentXEnd = xend;
  ord_t currentYStart = ystart;
  ord_t currentYEnd = yend;
  for(size_t j = 0, jend = depth; j != jend; ++j) {
    const location_t bit1 = thisPoint->y > (currentYStart + (currentYEnd - currentYStart) / 2);
    const location_t bit2 = thisPoint->x > (currentXStart + (currentXEnd - currentXStart) / 2);
    const location_t currentPosBits = (bit1 << 1) | bit2;
    locations[i] = (locations[i] << 2) | currentPosBits;

    const ord_t newWidth = (currentXEnd - currentXStart) / 2;
    currentXStart = floor((thisPoint->x - currentXStart) / newWidth) * newWidth + currentXStart;
    currentXEnd = currentXStart + newWidth;
    const ord_t newHeight = (currentYEnd - currentYStart) / 2;
    currentYStart = floor((thisPoint->y - currentYStart) / newHeight) * newHeight + currentYStart;
    currentYEnd = currentYStart + newHeight;
  }
}

struct linear_quadtree lqt_create_cuda(struct lqt_point* points, size_t len, 
                                       ord_t xstart, ord_t xend, 
                                       ord_t ystart, ord_t yend,
                                       size_t* depth) {
  // debug
  size_t cuda_mem_free = 0;
  size_t cuda_mem_total = 0;
  CubDebugExit(hipMemGetInfo(&cuda_mem_free, &cuda_mem_total));

  const size_t array_size = (sizeof(struct lqt_point) + sizeof(location_t)) * len * 2; // *2 for double-buffers
  const size_t num_blocks = array_size / cuda_mem_free + 1;
  printf("num blocks: %lu\n", num_blocks); // debug
  const size_t array_block_size = array_size / num_blocks;
  printf("free: %lu\tarray: %lu\tblocks: %lu\tblock size: %lu\n", cuda_mem_free, array_size, num_blocks, array_block_size); // debug
  
  const size_t block_len = len / num_blocks + (len % num_blocks != 0 ? 1 : 0);
  struct linear_quadtree* array_blocks = (struct linear_quadtree*) malloc(num_blocks * sizeof(linear_quadtree));

  for(size_t i = 0, end = num_blocks; i != end; ++i) {
    array_blocks[i].length = block_len;
    if(block_len * i + block_len  > len)
      array_blocks[i].length -= block_len * num_blocks - len; // fix the last block overlap
    array_blocks[i].points = (struct lqt_point*) malloc(sizeof(struct lqt_point) * array_blocks[i].length);
    memcpy(array_blocks[i].points, points + block_len * i, array_blocks[i].length * sizeof(struct lqt_point));
    array_blocks[i] = lqt_sortify_cuda_mem(lqt_nodify_cuda_mem(array_blocks[i].points, array_blocks[i].length, xstart, xend, ystart, yend, depth));
  }
  
  struct linear_quadtree lqt = lqt_merge(array_blocks, num_blocks, points, len);
  for(size_t i = 0, end = num_blocks; i != end; ++i)
    lqt_delete(array_blocks[i]);
  free(array_blocks);
  return lqt;
}

/// unnecessarily allocates and frees CUDA memory twice
struct linear_quadtree lqt_create_cuda_slow(struct lqt_point* points, size_t len, 
                                       ord_t xstart, ord_t xend, 
                                       ord_t ystart, ord_t yend,
                                       size_t* depth) {
  return lqt_sortify_cuda(lqt_nodify_cuda(points, len, xstart, xend, ystart, yend, depth));
}


struct linear_quadtree lqt_nodify_cuda(struct lqt_point* points, size_t len, 
                                       ord_t xstart, ord_t xend, 
                                       ord_t ystart, ord_t yend,
                                       size_t* depth) {
  *depth = LINEAR_QUADTREE_DEPTH;

  const size_t THREADS_PER_BLOCK = 512;

  location_t*       cuda_locations;
  struct lqt_point* cuda_points;

  hipMalloc((void**)&cuda_locations, len * sizeof(location_t));
  hipMalloc((void**)&cuda_points, len * sizeof(struct lqt_point));
  hipMemcpy(cuda_points, points, len * sizeof(struct lqt_point), hipMemcpyHostToDevice);
  hipMemset(cuda_locations, 0, len * sizeof(location_t)); // debug
  nodify_kernel<<<(len + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(cuda_points, cuda_locations, *depth, xstart, xend, ystart, yend, len);
  location_t* locations = (location_t*) malloc(len * sizeof(location_t));
  hipMemcpy(locations, cuda_locations, len * sizeof(location_t), hipMemcpyDeviceToHost);
  hipFree(cuda_locations);
  hipFree(cuda_points);

  struct linear_quadtree lqt;
  lqt.points    = points;
  lqt.locations = locations;
  lqt.length    = len;
  return lqt;
}

struct linear_quadtree lqt_sortify_cuda(struct linear_quadtree lqt) {
  DoubleBuffer<location_t> d_keys;
  DoubleBuffer<lqt_point> d_values;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(location_t) * lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(location_t) * lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[0], sizeof(lqt_point) * lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(lqt_point) * lqt.length));


  CubDebugExit( hipMemcpy(d_keys.d_buffers[0], lqt.locations, sizeof(location_t) * lqt.length, hipMemcpyHostToDevice));
  CubDebugExit( hipMemcpy(d_values.d_buffers[0], lqt.points, sizeof(lqt_point) * lqt.length, hipMemcpyHostToDevice));

  size_t temp_storage_bytes = 0;
  void* d_temp_storage = NULL;
  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, lqt.length));
  CubDebugExit( g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, lqt.length));
  
  CubDebugExit( hipMemcpy(lqt.locations, d_keys.Current(), lqt.length * sizeof(location_t), hipMemcpyDeviceToHost));
  CubDebugExit( hipMemcpy(lqt.points, d_values.Current(), lqt.length * sizeof(lqt_point), hipMemcpyDeviceToHost));

  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_temp_storage));
  return lqt;
}

void print_array_uint(unsigned int* array, const size_t len) {
  if(len == 0)
    return;
  printf("[%u", array[0]);
  for(size_t i = 1, end = len; i != end; ++i)
    printf(" %u", array[i]);
  printf("]");
}
void print_array_int(int* array, const size_t len) {
  if(len == 0)
    return;
  printf("[%d", array[0]);
  for(size_t i = 1, end = len; i != end; ++i)
    printf(" %d", array[i]);
  printf("]");
}

template <typename T> struct fmt_traits;
template <>
struct fmt_traits<int> {
  static const char* str() {return "%d";}
};
template <>
struct fmt_traits<unsigned int> {
  static const char* str() {return "%u";}
};
template <>
struct fmt_traits<location_t> {
  static const char* str() {return "%lu";}
};

template <typename T>
void print_array(T* array, const size_t len) {
  if(len == 0)
    return;
  printf("[");
  printf(fmt_traits<T>::str(), array[0]);
  for(size_t i = 1, end = len; i != end; ++i) {
    printf(" ");
    printf(fmt_traits<T>::str(), array[i]);
  }
  printf("]");
}

// @return CUDA-allocated points and locations, along with existing host-allocated points
struct linear_quadtree_cuda lqt_nodify_cuda_mem(struct lqt_point* points, size_t len, 
                                                ord_t xstart, ord_t xend, 
                                                ord_t ystart, ord_t yend,
                                                size_t* depth) {
  const size_t THREADS_PER_BLOCK = 512;
  *depth = LINEAR_QUADTREE_DEPTH;
  location_t*       cuda_locations;
  struct lqt_point* cuda_points;

  CubDebugExit(g_allocator.DeviceAllocate((void**)&cuda_locations, sizeof(location_t) * len));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&cuda_points, sizeof(lqt_point) * len));
//  hipMalloc((void**)&cuda_locations, len * sizeof(location_t));
//  hipMalloc((void**)&cuda_points, len * sizeof(struct lqt_point));
  hipMemcpy(cuda_points, points, len * sizeof(struct lqt_point), hipMemcpyHostToDevice);
  hipMemset(cuda_locations, 0, len * sizeof(location_t)); // debug
  nodify_kernel<<<(len + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(cuda_points, cuda_locations, *depth, xstart, xend, ystart, yend, len);

  struct linear_quadtree_cuda lqt;
  lqt.points         = points;
  lqt.cuda_locations = cuda_locations;
  lqt.cuda_points    = cuda_points;
  lqt.length         = len;
  return lqt;
}


struct linear_quadtree lqt_sortify_cuda_mem(struct linear_quadtree_cuda cuda_lqt) {
  //  printf("DEBUG lqt_sortify_cuda_mem\n"); // debug

  DoubleBuffer<location_t> d_keys;
  DoubleBuffer<lqt_point> d_values;
  d_keys.d_buffers[0]   = cuda_lqt.cuda_locations; // reuse the nodify CUDA memory for the cub buffers
  d_values.d_buffers[0] = cuda_lqt.cuda_points;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(location_t) * cuda_lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(lqt_point) * cuda_lqt.length));

  size_t temp_storage_bytes = 0;
  void* d_temp_storage = NULL;
  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, cuda_lqt.length));
  //  printf("temp storage: %lu\n", temp_storage_bytes);  // debug
  CubDebugExit( g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, cuda_lqt.length));

  struct linear_quadtree lqt;
  lqt.length = cuda_lqt.length;
  lqt.locations = (location_t*) malloc(lqt.length * sizeof(location_t));
  CubDebugExit( hipMemcpy(lqt.locations, d_keys.Current(), lqt.length * sizeof(location_t), hipMemcpyDeviceToHost));
  lqt.points = cuda_lqt.points;
  CubDebugExit( hipMemcpy(lqt.points, d_values.Current(), lqt.length * sizeof(lqt_point), hipMemcpyDeviceToHost));

  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_temp_storage));
  return lqt;
}
